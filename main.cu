#include <iostream>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <ctime>

using namespace std;
__global__ void AddInts(int *a,int *b,int count) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < count)
        a[id] += b[id];
}
int main()
{

    // tao random seed bang cach su dung timer
    srand(time(0));
    int count = 100;
    int *h_a = new int[count];
    int *h_b = new int[count];
    for (int i = 0;i < count;i++) {
        h_a[i] = rand() % 1000;
        h_b[i] = rand() % 1000;
    }
    cout << "truoc khi cong: " << endl;
    for (int i  =0 ;i < 5;i++) {
        cout << h_a[i] << " " << h_b[i] << endl;
    }
    // chuyen bo nho tu host sang device
    int *d_a,*d_b;
    if (hipMalloc(&d_a,sizeof(int) * count) != hipSuccess) {
        cout << "That bai de cap phat bo nho";
        // Neu cap phat that bai thi giai phong bo nho vua moi cap phat
        hipFree(d_a);
        return 0;
    }
    if (hipMalloc(&d_b,sizeof(int) * count) != hipSuccess) {
        cout << "That bai de cap phat bo nho";
        hipFree(&d_b);
        return 0;
    }

    if (hipMemcpy(d_a,h_a,sizeof(int) * count,hipMemcpyHostToDevice) != hipSuccess) {
        cout << "that bai de copy";
        hipFree(d_a);
        hipFree(d_b);
        return 0;
    }
    if (hipMemcpy(d_b,h_b,sizeof(int) * count,hipMemcpyHostToDevice) != hipSuccess) {
        cout << "that bai de copy";
        hipFree(d_a);
        hipFree(d_b);
        return 0;
    }
    AddInts<<<count / 256 + 1,256>>>(d_a,d_b,count);
    if (hipMemcpy(h_a,d_a,sizeof(int) * count,hipMemcpyDeviceToHost) != hipSuccess) {
        delete[] h_a;
        delete[] h_b;
        hipFree(d_a);
        hipFree(d_b);
        return 0;
    }
    for (int i = 0;i < 5;i++) {
        //cout << "It's" << h_a[i];
        printf("%d ",h_a[i]);
    }
    delete[] h_a;
    delete[] h_b;
}