#include <iostream>
#include <hip/hip_runtime.h>



using namespace std;
__device__ float CalcDistance(float3 a, float3 b) {
    return (a.x - b.x) * (a.x - b.x)   + (a.y - b.y) * (a.y - b.y)  + (a.z  - b.z) * (a.z -b.z);
}
__global__ void FindClosest(float3* point,int* id,int count) {
    if (count <= 1)
        return;

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= count)
        return;
    float3 thispoint = point[idx];
    float smallestSoFar = 3.4e38;

    for (int i = 0;i < count;i++) {
        if (i == idx) continue;

        auto dis = CalcDistance(thispoint,point[i]);
        if (dis < smallestSoFar) {
            smallestSoFar = dis;
            id[idx] = i;
        }
    }
}
int main()
{
    float3* point;
    int *id;
    int count;


    count = 5;
    hipMallocManaged(&point,sizeof(float3) * count);
    hipMallocManaged(&id,sizeof(int) * count);

    for (int i = 0;i < count;i++) {
        point[i].x = rand() % 10;
        point[i].y = rand() % 10;
        point[i].z = rand() % 10;
    }
    FindClosest<<<count / 255 + 1,255>>>(point,id,count);
    hipDeviceSynchronize();
    for (int i =0 ;i < count;i++)
        cout << "Point: " << point[i].x << ' ' << point[i].y << ' ' << point[i].z << " Neareast Neighbor index: " << id[i] << '\n';
}